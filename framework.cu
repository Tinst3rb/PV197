#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "kernel.cu"
#include "kernel_CPU.C"

int main(int argc, char **argv){
    // CPU data
    float *input, *average, *average_gpu;
    input = average = NULL;
    // GPU counterparts
    float *dinput, *daverage;
    dinput = daverage = NULL;

    // parse command line
    int device = 0;
    if (argc == 2) 
        device = atoi(argv[1]);
    if (hipSetDevice(device) != hipSuccess){
        fprintf(stderr, "Cannot set CUDA device!\n");
        exit(1);
    }
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Using device %d: \"%s\"\n", device, deviceProp.name);

    // create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate and set host memory
    input = (float*)malloc(SIZE*sizeof(input[0]));
    average = (float*)malloc(SIZE*sizeof(average[0]));
    average_gpu = (float*)malloc(SIZE*sizeof(average[0]));
    for (int i = 0; i < SIZE; i++)
        input[i] = (float)rand() / float(RAND_MAX);
 
    // allocate and set device memory
    if (hipMalloc((void**)&dinput, SIZE*sizeof(dinput[0])) != hipSuccess
    || hipMalloc((void**)&daverage, SIZE*sizeof(daverage[0])) != hipSuccess){
        fprintf(stderr, "Device memory allocation error!\n");
        goto cleanup;
    }
    hipMemcpy(dinput, input, SIZE*sizeof(dinput[0]), hipMemcpyHostToDevice);

    // solve on CPU
    printf("Solving on CPU...\n");
    hipEventRecord(start, 0);
    solveCPU(average, input, SIZE);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("CPU performance: %f megavalues/s\n",
        float(SIZE)/time/1e3f);

    // solve on GPU
    printf("Solving on GPU...\n");
    solveGPU(daverage, dinput, SIZE);
    hipEventRecord(start, 0);
    // for(int i =0; i < 100; i++) 
        solveGPU(daverage, dinput, SIZE);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("GPU performance: %f megavalues/s\n",
        float(SIZE)/time/1e3f);

    // check GPU results
    hipMemcpy(average_gpu, daverage, SIZE*sizeof(average_gpu[0]), hipMemcpyDeviceToHost);
    for (int i = 0; i < SIZE; i++)
        if ((average_gpu[i] != average_gpu[i]) /* catches NaN */
	|| (std::abs(average[i]-average_gpu[i]) > 0.0001f)) { 
            fprintf(stderr, "Data mismatch at index %i: %f should be %f :-(\n", i, average_gpu[i], average[i]);
            goto cleanup;
        }
    printf("Test OK.\n");

cleanup:
    hipEventDestroy(start);
    hipEventDestroy(stop);

    if (daverage) hipFree(daverage);
    if (dinput) hipFree(dinput);
    if (average) free(average);
    if (input) free(input);
    if (average_gpu) free(average_gpu);

    return 0;
}
